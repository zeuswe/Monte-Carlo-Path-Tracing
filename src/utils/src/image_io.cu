#include "hip/hip_runtime.h"
#include "image_io.cuh"

#include <algorithm>
#include <array>
#include <set>

#include <cmath>
#include <cstdio>
#include <tinyexr.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_IMAGE_RESIZE_IMPLEMENTATION
extern "C"
{
#include "stb_image.h"
#include "stb_image_resize2.h"
#include "stb_image_write.h"
}

#include "misc.cuh"

namespace csrt
{

void image_io::Write(const int width, const int height,
                     const float *frame_buffer, const std::string &filename)
{
    unsigned char *color = new unsigned char[width * height * 3];
    int offset = 0;
    for (int j = 0; j < height; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            offset = (j * width + i) * 3;
            color[offset] = static_cast<unsigned char>(
                std::min(255, static_cast<int>(255 * frame_buffer[offset])));
            color[offset + 1] = static_cast<unsigned char>(std::min(
                255, static_cast<int>(255 * frame_buffer[offset + 1])));
            color[offset + 2] = static_cast<unsigned char>(std::min(
                255, static_cast<int>(255 * frame_buffer[offset + 2])));
        }
    }

    int ret =
        stbi_write_png(filename.c_str(), width, height, 3, color, width * 3);
    delete[] color;
    color = nullptr;
    if (ret == 0)
    {
        fprintf(stderr, "[error] write image failed.\n");
    }
    else
    {
        fprintf(stderr, "[info] save result as image \"%s\".\n",
                filename.c_str());
    }
}

void image_io::Read(const std::string &filename, const float gamma,
                    const int *width_max, int *width, int *height, int *channel,
                    std::vector<float> *data)
{
    const std::set<std::string> support_format = {
        "EXR", "exr", "jpg", "jpeg", "JPG", "JPEG", "png", "PNG",
        "tga", "TGA", "bmp", "BMP",  "psd", "PSD",  "gif", "GIF",
        "hdr", "HDR", "pic", "PIC",  "pgm", "PGM",  "ppm", "PPM"};
    std::string suffix = GetSuffix(filename);
    if (!support_format.count(suffix))
    {
        fprintf(stderr, "[error] unsupport input image format for image '%s\n'",
                filename.c_str());
        exit(1);
    }

    float *raw_data = nullptr;
    switch (Hash(suffix.c_str()))
    {
    case "exr"_hash:
    case "EXR"_hash:
    {
        const char *err = nullptr;
        if (LoadEXR(&raw_data, width, height, filename.c_str(), &err) !=
            TINYEXR_SUCCESS)
        {
            fprintf(stderr, "[error] load image '%s' failed.",
                    filename.c_str());
            if (err)
            {
                fprintf(stderr, "\t%s", err);
                FreeEXRErrorMessage(err);
            }
            exit(1);
        }
        if (gamma != 0.0f)
        {
            int num_component = *width * *height * *channel;
            for (int i = 0; i < num_component; ++i)
                raw_data[i] = std::pow(raw_data[i], gamma);
        }
        *channel = 4;
        break;
    }
    default:
    {
        stbi_uc *raw_data_uc =
            stbi_load(filename.c_str(), width, height, channel, 0);
        if (raw_data_uc == nullptr)
        {
            fprintf(stderr, "[error] load image '%s' failed.",
                    filename.c_str());
            exit(1);
        }
        int num_component = *width * *height * *channel;
        raw_data = new float[num_component];
        if (suffix != "HDR" && suffix != "hdr")
        {
            for (int i = 0; i < num_component; ++i)
                raw_data[i] = static_cast<int>(raw_data_uc[i]) / 255.0f;
            if (gamma == 0.0f || gamma == -1.0f)
            {
                for (int i = 0; i < num_component; ++i)
                {
                    raw_data[i] =
                        raw_data[i] <= 0.04045f
                            ? raw_data[i] / 12.92f
                            : std::pow((raw_data[i] + 0.055f) / 1.055f, 2.4f);
                }
            }
            else
            {
                for (int i = 0; i < num_component; ++i)
                    raw_data[i] = std::pow(raw_data[i], gamma);
            }
        }
        else
        {
            for (int i = 0; i < num_component; ++i)
                raw_data[i] = static_cast<int>(raw_data_uc[i]);

            if (gamma == -1.0f)
            {
                for (int i = 0; i < num_component; ++i)
                {
                    raw_data[i] =
                        raw_data[i] <= 0.04045f
                            ? raw_data[i] / 12.92f
                            : std::pow((raw_data[i] + 0.055f) / 1.055f, 2.4f);
                }
            }
            else if (gamma != 0.0f)
            {
                for (int i = 0; i < num_component; ++i)
                    raw_data[i] = std::pow(raw_data[i], gamma);
            }
        }

        stbi_image_free(raw_data_uc);
        break;
    }
    }

    if (width_max != nullptr && *width > *width_max)
    {
        int height_target = *width_max * *height / *width;
        float *target_data = new float[*width_max * height_target * *channel];
        Resize(raw_data, *width, *height, 0, target_data, *width_max,
               height_target, 0, *channel);
        raw_data = target_data;
        *width = *width_max;
        *height = height_target;
    }

    *data =
        std::vector<float>(raw_data, raw_data + *width * *height * *channel);
    SAFE_DELETE_ARRAY(raw_data);
}

void image_io::Resize(const float *input_pixels, int input_w, int input_h,
                      int input_stride_in_bytes, float *output_pixels,
                      int output_w, int output_h, int output_stride_in_bytes,
                      int num_channels)
{
    stbir_resize_float_linear(input_pixels, input_w, input_h,
                              input_stride_in_bytes, output_pixels, output_w,
                              output_h, output_stride_in_bytes,
                              static_cast<stbir_pixel_layout>(num_channels));
}

} // namespace csrt