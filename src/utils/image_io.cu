#include "hip/hip_runtime.h"
#include "image_io.cuh"

#include <set>
#include <algorithm>
#include <array>

#include <cstdio>
#include <tinyexr.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_IMAGE_RESIZE_IMPLEMENTATION
extern "C"
{
#include <stb_image.h>
#include <stb_image_write.h>
#include <stb_image_resize.h>
}

#include "misc.cuh"

void image_io::Write(int width, int height, float *frame_buffer, const std::string &filename)
{
    unsigned char *color = new unsigned char[width * height * 3];
    int offset = 0;
    for (int j = 0; j < height; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            offset = (j * width + i) * 3;
            color[offset] = static_cast<unsigned char>(std::min(
                255, static_cast<int>(255 * frame_buffer[offset])));
            color[offset + 1] = static_cast<unsigned char>(std::min(
                255, static_cast<int>(255 * frame_buffer[offset + 1])));
            color[offset + 2] = static_cast<unsigned char>(std::min(
                255, static_cast<int>(255 * frame_buffer[offset + 2])));
        }
    }

    int ret = stbi_write_png(filename.c_str(), width, height, 3, color, width * 3);
    delete[] color;
    color = nullptr;
    if (ret == 0)
    {
        fprintf(stderr, "[error] write image failed.\n");
    }
    else
    {
        fprintf(stderr, "[info] save result as image \"%s\".", filename.c_str());
    }
}

void image_io::Read(const std::string &filename, float gamma, int *width, int *height, int *channel,
                    std::vector<float> *data, int *width_max)
{
    const std::set<std::string> support_format = {"EXR", "exr",
                                                  "jpg", "jpeg", "JPG", "JPEG",
                                                  "png", "PNG",
                                                  "tga", "TGA",
                                                  "bmp", "BMP",
                                                  "psd", "PSD",
                                                  "gif", "GIF",
                                                  "hdr", "HDR",
                                                  "pic", "PIC",
                                                  "pgm", "PGM", "ppm", "PPM"};
    std::string suffix = GetSuffix(filename);
    if (!support_format.count(suffix))
    {
        fprintf(stderr, "[error] unsupport input image format for image '%s\n'",
                filename.c_str());
        exit(1);
    }

    float *raw_data = nullptr;
    switch (Hash(suffix.c_str()))
    {
    case "exr"_hash:
    case "EXR"_hash:
    {
        const char *err = nullptr;
        if (LoadEXR(&raw_data, width, height, filename.c_str(), &err) != TINYEXR_SUCCESS)
        {
            fprintf(stderr, "[error] load image '%s' failed.", filename.c_str());
            if (err)
            {
                fprintf(stderr, "\t%s", err);
                FreeEXRErrorMessage(err);
            }
            exit(1);
        }
        *channel = 4;
        break;
    }
    default:
    {
        stbi_uc *raw_data_uc = stbi_load(filename.c_str(), width, height, channel, 0);
        if (raw_data_uc == nullptr)
        {
            fprintf(stderr, "[error] load image '%s' failed.", filename.c_str());
            exit(1);
        }
        int num_component = *width * *height * *channel;
        raw_data = new float[num_component];
        for (int i = 0; i < num_component; ++i)
            raw_data[i] = static_cast<int>(raw_data_uc[i]) / 255.0f;

        if (suffix != "HDR" && suffix != "hdr" && suffix != "EXR" && suffix != "exr")
        {
            if (gamma == -1.0f)
            {
                for (int i = 0; i < num_component; ++i)
                {
                    if (raw_data[i] <= 0.04045f)
                        raw_data[i] = raw_data[i] / 12.92f;
                    else
                        raw_data[i] = std::pow((raw_data[i] + 0.055f) / 1.055f, 2.4f);
                }
            }
            else
            {
                for (int i = 0; i < num_component; ++i)
                    raw_data[i] = std::pow(raw_data[i], gamma);
            }
        }
        else if (gamma != -1.0f)
        {
            for (int i = 0; i < num_component; ++i)
                raw_data[i] = std::pow(raw_data[i], gamma);
        }
        stbi_image_free(raw_data_uc);
        break;
    }
    }

    if (width_max != nullptr && *width > *width_max)
    {
        int height_target = *width_max * *height / *width;
        float *target_data = new float[*width_max * height_target * *channel];
        stbir_resize_float(raw_data, *width, *height, 0, target_data, *width_max,
                           height_target, 0, *channel);
        raw_data = target_data;
        *width = *width_max;
        *height = height_target;
    }

    *data = std::vector<float>(raw_data, raw_data + *width * *height * *channel);
    SAFE_DELETE_ARRAY(raw_data);
}
