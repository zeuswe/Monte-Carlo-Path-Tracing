#include "hip/hip_runtime.h"
#include "tensor/vec2.cuh"

#include <cmath>

NAMESPACE_BEGIN(rt)

QUALIFIER_DEVICE Vec2 &Vec2::operator+=(const Vec2 &vec)
{
    u += vec.u, v += vec.v;
    return *this;
}

QUALIFIER_DEVICE Vec2 &Vec2::operator-=(const Vec2 &vec)
{
    u -= vec.u, v -= vec.v;
    return *this;
}

QUALIFIER_DEVICE Vec2 &Vec2::operator*=(const Vec2 &vec)
{
    u *= vec.u, v *= vec.v;
    return *this;
}

QUALIFIER_DEVICE Vec2 &Vec2::operator/=(const Vec2 &vec)
{
    const float k0 = 1.0f / vec.u, k1 = 1.0f / vec.v;
    u *= k0, v *= k1;
    return *this;
}

QUALIFIER_DEVICE Vec2 &Vec2::operator*=(const float t)
{
    u *= t, v *= t;
    return *this;
}

QUALIFIER_DEVICE Vec2 &Vec2::operator/=(const float t)
{
    const float k = 1.0f / t;
    u *= k, v *= k;
    return *this;
}

QUALIFIER_DEVICE float Vec2::Length()
{
    return sqrtf(u * u + v * v);
}

QUALIFIER_DEVICE Vec2 Vec2::Normalize()
{
    const float k = 1.0f / Length();
    return *this * k;
}

QUALIFIER_DEVICE Vec2 operator+(const Vec2 &v1, const Vec2 &v2)
{
    return {v1.u + v2.u, v1.v + v2.v};
}

QUALIFIER_DEVICE Vec2 operator-(const Vec2 &v1, const Vec2 &v2)
{
    return {v1.u - v2.u, v1.v - v2.v};
}

QUALIFIER_DEVICE Vec2 operator*(const Vec2 &v1, const Vec2 &v2)
{
    return {v1.u * v2.u, v1.v * v2.v};
}

QUALIFIER_DEVICE Vec2 operator/(const Vec2 &v1, const Vec2 &v2)
{
    const float k0 = 1.0f / v2.u, k1 = 1.0f / v2.v;
    return {v1.u * k0, v1.v * k1};
}

QUALIFIER_DEVICE Vec2 operator+(const Vec2 &vec, float t)
{
    return {vec.u + t, vec.v + t};
}

QUALIFIER_DEVICE Vec2 operator-(const Vec2 &vec, float t)
{
    return {vec.u - t, vec.v - t};
}

QUALIFIER_DEVICE Vec2 operator*(const Vec2 &vec, float t)
{
    return {vec.u * t, vec.v * t};
}

QUALIFIER_DEVICE Vec2 operator/(const Vec2 &vec, float t)
{
    const float k = 1.0f / t;
    return {vec.u * k, vec.v * k};
}

QUALIFIER_DEVICE Vec2 operator+(float t, const Vec2 &vec)
{
    return {t + vec.u, t + vec.v};
}

QUALIFIER_DEVICE Vec2 operator-(float t, const Vec2 &vec)
{
    return {t - vec.u, t - vec.v};
}

QUALIFIER_DEVICE Vec2 operator*(float t, const Vec2 &vec)
{
    return {t * vec.u, t * vec.v};
}

QUALIFIER_DEVICE Vec2 operator/(float t, const Vec2 &vec)
{
    const float k0 = 1.0f / vec.u, k1 = 1.0f / vec.v;
    return {t * k0, t * k1};
}

QUALIFIER_DEVICE float Dot(const Vec2 &v1, const Vec2 &v2)
{
    return v1.u * v2.u + v1.v * v2.v;
}

QUALIFIER_DEVICE Vec2 Min(const Vec2 &v1, const Vec2 &v2)
{
    return {fminf(v1.u, v2.u), fminf(v1.v, v2.v)};
}

QUALIFIER_DEVICE Vec2 Max(const Vec2 &v1, const Vec2 &v2)
{
    return {fmaxf(v1.u, v2.u), fmaxf(v1.v, v2.v)};
}

NAMESPACE_END(rt)