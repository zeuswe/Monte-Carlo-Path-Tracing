#include "hip/hip_runtime.h"
#include "csrt/renderer/textures/checkerboard.cuh"

namespace csrt
{

QUALIFIER_D_H Vec3 GetColorCheckerboard(const CheckerboardData &data,
                                        const Vec2 &texcoord)
{
    Vec3 uv = TransformPoint(data.to_uv, {texcoord, 0.0f});
    while (uv.x > 1)
        uv.x -= 1;
    while (uv.x < 0)
        uv.x += 1;
    while (uv.y > 1)
        uv.y -= 1;
    while (uv.y < 0)
        uv.y += 1;
    const int x = 2 * static_cast<int>(static_cast<int>(uv.x * 2) % 2) - 1,
              y = 2 * static_cast<int>(static_cast<int>(uv.y * 2) % 2) - 1;
    return (x * y == 1) ? data.color0 : data.color1;
}

QUALIFIER_D_H Vec2 GetGradientCheckerboard(const CheckerboardData &data,
                                           const Vec2 &texcoord)
{
    constexpr float delta = 1e-4f, norm = 1.0f / delta;
    const float value = Length(GetColorCheckerboard(data, texcoord)),
                value_u = Length(
                    GetColorCheckerboard(data, texcoord + Vec2{delta, 0})),
                value_v = Length(
                    GetColorCheckerboard(data, texcoord + Vec2{0, delta}));
    return {(value_u - value) * norm, (value_v - value) * norm};
}

QUALIFIER_D_H bool IsTransparentCheckerboard(const CheckerboardData &data,
                                             const Vec2 &texcoord,
                                             uint32_t *seed)
{
    return false;
}

} // namespace csrt